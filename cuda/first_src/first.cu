#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello, CUDA!\n");
}

void checkCudaError(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " (" << hipGetErrorString(result) << ")" << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    kernel<<<1, 1>>>();
    checkCudaError(hipGetLastError(), "Kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Device synchronization");
    return 0;
}

/**
    * Compile with:
    *  nvcc -o first first.cu
    * Run with:
    * ./first
    * Expected output:
    * Hello, CUDA!
 */
